﻿#include "cuda_includes.cuh"

#include <stdlib.h>
#include <stdio.h>

void HandleError(hipError_t err, const char* file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
		exit(EXIT_FAILURE);
	}
}

struct Sphere;